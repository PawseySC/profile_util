#include "hip/hip_runtime.h"
#include "kernels.h"
#include "common.h"

template <typename T> __global__ 
void vector_square(const T *A_d, T *C_d, size_t N)
{
    size_t offset = (blockIdx.x * blockDim.x + threadIdx.x);
    size_t stride = blockDim.x * gridDim.x;

    for (size_t i=offset; i<N; i+=stride) {
        C_d[i] = A_d[i] * A_d[i];
    }
}

void compute_kernel1(size_t N, 
    std::vector<int*> &x_int_gpu, 
    std::vector<int*> &y_int_gpu, 
    std::vector<float*> &x_float_gpu, 
    std::vector<float*> &y_float_gpu, 
    std::vector<double*> &x_double_gpu, 
    std::vector<double*> &y_double_gpu,
    size_t blocksize, 
    size_t threadsperblock
    ) 
{
    int nDevices;
    pu_gpuErrorCheck(pu_gpuGetDeviceCount(&nDevices));
    for (auto idev=0;idev<nDevices;idev++) {
        LocalLogger()<<" at device "<<idev<<" and doing stuff "<<std::endl;
        pu_gpuErrorCheck(pu_gpuSetDevice(idev));

        auto time_kernel = NewTimer();
        pu_gpuLaunchKernel(vector_square, 
            dim3(blocksize), dim3(threadsperblock), 0, 0,
            x_int_gpu[idev], y_int_gpu[idev], N);
        pu_gpuLaunchKernel(vector_square, 
            dim3(blocksize), dim3(threadsperblock), 0, 0,
            x_float_gpu[idev], y_float_gpu[idev], N);
        pu_gpuLaunchKernel(vector_square, 
            dim3(blocksize), dim3(threadsperblock), 0, 0,
            x_double_gpu[idev], y_double_gpu[idev], N);
        LogTimeTakenOnDevice(time_kernel);
    }
}

// __global__ void vector_square(const int *, int *, size_t);
// __global__ void vector_square(const float *, float *, size_t);
// __global__ void vector_square(const double *, double *, size_t);
